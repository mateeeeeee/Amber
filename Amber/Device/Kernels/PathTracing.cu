#include "hip/hip_runtime.h"
#pragma once
#include "DeviceCommon.cuh"
#include "Device/DeviceHostCommon.h"
#include "Random.cuh"
#include "Color.cuh"
#include "ONB.cuh"
#include "Disney.cuh"

using namespace amber;

extern "C" 
{
	__constant__ LaunchParams params;
}

template<typename T>
__forceinline__ __device__ T Interpolate(T const& t0, T const& t1, T const& t2, float2 bary)
{
	return t0 * (1.0f - bary.x - bary.y) + bary.x * t1 + bary.y * t2;
}

__device__ __forceinline__ void UnpackMaterial(DisneyMaterial& mat_params, Uint32 id, float2 uv)
{
	MaterialGPU material = params.materials[id];
	if (material.diffuse_tex_id >= 0)
	{
		float4 sampled = tex2D<float4>(params.textures[material.diffuse_tex_id], uv.x, uv.y);
		mat_params.base_color = material.base_color * make_float3(sampled.x, sampled.y, sampled.z);
	}
	else
	{
		mat_params.base_color = material.base_color;
	}

	if (material.emissive_tex_id >= 0)
	{
		float4 sampled = tex2D<float4>(params.textures[material.emissive_tex_id], uv.x, uv.y);
		mat_params.emissive = material.emissive_color * make_float3(sampled.x, sampled.y, sampled.z);
	}
	else
	{
		mat_params.emissive = material.emissive_color;
	}

	if (material.metallic_roughness_tex_id >= 0)
	{
		float4 sampled = tex2D<float4>(params.textures[material.metallic_roughness_tex_id], uv.x, uv.y);
		mat_params.ao = sampled.x;
		mat_params.roughness = sampled.y * material.roughness;
		mat_params.metallic = sampled.z * material.metallic;
	}
	else
	{
		mat_params.ao = 1.0f;
		mat_params.roughness = material.roughness;
		mat_params.metallic = material.metallic;
	}

	if (material.normal_tex_id >= 0)
	{
		float4 sampled = tex2D<float4>(params.textures[material.normal_tex_id], uv.x, uv.y);
		mat_params.normal = make_float3(sampled.x, sampled.y, sampled.z);
		mat_params.normal = 2.0f * mat_params.normal - 1.0f;
	}
	else
	{
		mat_params.normal = make_float3(0.0f, 0.0f, 1.0f);
	}

	mat_params.specular_tint = material.specular_tint;
	mat_params.anisotropy = material.anisotropy;
	mat_params.sheen = material.sheen;
	mat_params.sheen_tint = material.sheen_tint;
	mat_params.clearcoat = material.clearcoat;
	mat_params.clearcoat_gloss = material.clearcoat_gloss;
	mat_params.ior = material.ior;
	mat_params.specular_transmission = material.specular_transmission;
}

__device__ __forceinline__ float3 ApplyNormalMap(const float3& normal_map, const float3& v_x, const float3& v_y, const float3& v_z)
{
	return normalize(normal_map.x * v_x + normal_map.y * v_y + normal_map.z * v_z);
}

__device__ __forceinline__ float3 GetRayDirection(uint2 pixel, uint2 screen, unsigned int seed)
{
	float3 const U = params.cam_u;
	float3 const V = params.cam_v;
	float3 const W = params.cam_w;

	float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));
	float2 d = (make_float2(pixel) + subpixel_jitter) / make_float2(screen);
	d.y = 1.0f - d.y;
	d = 2.0f * d - 1.0f;

	float tan_half_fovy = tan(params.cam_fovy * 0.5f);
	float aspect_ratio = params.cam_aspect_ratio;

	float3 ray_direction = normalize(d.x * aspect_ratio * tan_half_fovy * U + d.y * tan_half_fovy * V + W);
	return ray_direction;
}

__device__ __forceinline__ float3 SampleDirectLight(DisneyMaterial const& mat_params, float3 const& hit_point, float3 const& w_o, OrthonormalBasis const& ort, Uint32& seed)
{
	Uint32 light_index = rnd(seed) * params.light_count;
	LightGPU light = params.lights[light_index];

	float3 const& v_x = ort.tangent;
	float3 const& v_y = ort.binormal;
	float3 const& v_z = ort.normal;

	float3 radiance = make_float3(0.0f);
	if (light.type == LightType_Directional)
	{
		float3 light_dir = normalize(light.direction);
		if (!TraceOcclusion(params.traversable, hit_point + M_EPSILON * v_z, -light_dir, M_EPSILON, M_INF))
		{
			float3 bsdf = DisneyBrdf(mat_params, v_z, w_o, -light_dir, v_x, v_y);
			radiance = bsdf * light.color * abs(dot(-light_dir, v_z));
		}

		float3 w_i;
		float bsdf_pdf;
		float3 bsdf = SampleDisneyBrdf(mat_params, v_z, w_o, v_x, v_y, seed, w_i, bsdf_pdf);

		if (length(bsdf) > M_EPSILON && bsdf_pdf >= M_EPSILON)
		{
			float light_pdf = 1.0f; 
			float w = PowerHeuristic(1.f, bsdf_pdf, 1.f, light_pdf);

			if (!TraceOcclusion(params.traversable, hit_point + M_EPSILON * v_z, w_i, M_EPSILON, M_INF))
			{
				float3 bsdf = DisneyBrdf(mat_params, v_z, w_o, -light_dir, v_x, v_y);
				radiance += bsdf * light.color * abs(dot(w_i, v_z)) * w / bsdf_pdf;
			}
		}
	}
	else if (light.type == LightType_Point)
	{
		float3 light_pos = light.position;
		float3 light_dir = light_pos - hit_point; 
		float dist = length(light_dir);
		light_dir = light_dir / dist;

		if (!TraceOcclusion(params.traversable, hit_point + M_EPSILON * v_z, light_dir, M_EPSILON, dist - M_EPSILON))
		{
			float attenuation = 1.0f / (dist * dist);
			float3 bsdf = DisneyBrdf(mat_params, v_z, w_o, light_dir, v_x, v_y);
			radiance = bsdf * light.color * abs(dot(light_dir, v_z)) * attenuation;
		}

		float3 w_i;
		float bsdf_pdf;
		float3 bsdf = SampleDisneyBrdf(mat_params, v_z, w_o, v_x, v_y, seed, w_i, bsdf_pdf);
		if (length(bsdf) > M_EPSILON && bsdf_pdf >= M_EPSILON)
		{
			float light_pdf = (dist * dist) / (abs(dot(light_dir, v_z)) * 1.0f); // light.radius);
			float w = PowerHeuristic(1.f, bsdf_pdf, 1.f, light_pdf);

			if (!TraceOcclusion(params.traversable, hit_point + M_EPSILON * v_z, w_i, M_EPSILON, dist - M_EPSILON))
			{
				float attenuation = 1.0f / (dist * dist); 
				radiance += bsdf * light.color * abs(dot(w_i, v_z)) * attenuation * w / bsdf_pdf;
			}
		}
	}
	return radiance;
}

__device__ void WriteToDenoiserBuffers(Uint32 idx, float3 const& albedo, float3 const& normal)
{
	if (params.denoiser_albedo != NULL)
	{
		params.denoiser_albedo[idx] = albedo;
	}

	if (params.denoiser_normals != NULL)
	{
		float3 view_normal;
		view_normal.x = dot(normal,  params.cam_u);
		view_normal.y = dot(normal,  params.cam_v);
		view_normal.z = dot(normal, -params.cam_w);
		params.denoiser_normals[idx] = view_normal;
	}
}

__device__ void WriteToDebugBuffer(Uint32 idx, float3 const& albedo, float3 const& normal, float2 const& uv, Uint32 material_id)
{
	if (params.output_type == PathTracerOutputGPU_Albedo)
	{
		params.debug_buffer[idx] = albedo;
		return;
	}
	if (params.output_type == PathTracerOutputGPU_Normal)
	{
		params.debug_buffer[idx] = normal;
		return;
	}
	if (params.output_type == PathTracerOutputGPU_UV)
	{
		params.debug_buffer[idx] = make_float3(uv, 0.0f);
		return;
	}
	if(params.output_type == PathTracerOutputGPU_MaterialID)
	{
		float3 material_id_color = make_float3(
			(material_id * 37) % 255 / 255.0, 
			(material_id * 59) % 255 / 255.0,
			(material_id * 97) % 255 / 255.0);
		params.debug_buffer[idx] = material_id_color;
		return;
	}
}


extern "C" 
__global__ void RG_NAME(rg)()
{
	OptixTraversableHandle scene = params.traversable;
	float3 const  eye = params.cam_eye;
	uint2  const  pixel  = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);
	uint2  const  screen = make_uint2(optixGetLaunchDimensions().x, optixGetLaunchDimensions().y);
	Uint32 samples = params.sample_count;
	Uint32 idx = pixel.x + pixel.y * screen.x;

	float3 radiance = make_float3(0.0f);
	float3 throughput = make_float3(1.0f);
	do
	{
		Uint32 seed = tea<4>(idx, samples + params.frame_index);
		float3 ray_origin = eye;
		float3 ray_direction = GetRayDirection(pixel, screen, seed);

		HitRecord hit_record{};
		hit_record.depth = 0;
		Uint32 p0 = PackPointer0(&hit_record), p1 = PackPointer1(&hit_record);

		for (Uint32 depth = 0; depth < params.max_depth; ++depth)
		{
			Trace(scene, ray_origin, ray_direction, M_EPSILON, M_INF, p0, p1);
			if (!hit_record.hit)
			{
				float3 const& dir = ray_direction;
				float u = (1.f + atan2(dir.x, -dir.z) * M_INV_PI) * 0.5f;
				float v = 1.0f - acos(dir.y) * M_INV_PI;
				float3 env_map_color = make_float3(0.0f);
				if (params.sky)
				{
					float4 sampled = tex2D<float4>(params.sky, u, v);
					env_map_color = make_float3(sampled.x, sampled.y, sampled.z);
				}

				radiance += env_map_color * throughput;

				if (depth == 0)
				{
					WriteToDenoiserBuffers(idx, make_float3(0.0f, 0.0f, 0.0f), make_float3(0.0f, 0.0f, 0.0f));
					WriteToDebugBuffer(idx, make_float3(0.0f, 0.0f, 0.0f), make_float3(0.0f, 0.0f, 0.0f), make_float2(0.0f, 0.0f), 0);
				}
				break;
			}

			DisneyMaterial material{};
			UnpackMaterial(material, hit_record.material_idx, hit_record.uv);

			float3 emissive = material.emissive;
			radiance += emissive * throughput;

			float3 w_o = -ray_direction;
			float3 v_x, v_y;
			float3 v_z = hit_record.N;
			if (material.specular_transmission == 0.0f && dot(w_o, v_z) < 0.0f)
			{
				v_z = -v_z;
			}

			OrthonormalBasis ort(v_z);
			v_x = ort.tangent;
			v_y = ort.binormal;

			//if (length(material.normal - make_float3(0.0f, 0.0f, 1.0f)) > 1e-4f)
			//{
			//	v_z = ApplyNormalMap(material.normal, v_x, v_y, v_z);
			//	ort = OrthonormalBasis(v_z);
			//	v_x = ort.tangent;
			//	v_y = ort.binormal;
			//}

			if (depth == 0)
			{
				WriteToDenoiserBuffers(idx, material.base_color, v_z);
				WriteToDebugBuffer(idx, material.base_color, v_z, hit_record.uv, hit_record.material_idx);
			}

			radiance += SampleDirectLight(material, hit_record.P, w_o, ort, seed) * throughput;

			float3 w_i;
			float pdf;
			float3 bsdf = SampleDisneyBrdf(material, v_z, w_o, v_x, v_y, seed, w_i, pdf);

			if (params.output_type == PathTracerOutputGPU_Custom && depth == 0)
			{
				bool entering = dot(w_o, v_z) > 0.f;
				float dot_wi_n = dot(w_i, v_z); // Sampled direction vs. normal
				bool is_reflected = SameHemisphere(w_o, w_i, v_z); // 1.0 if reflected, 0.0 if refracted
				params.debug_buffer[idx] = make_float3(dot_wi_n, is_reflected ? 1.0f : 0.0f, pdf);
				return;
			}

			if (pdf == 0.0f || length(bsdf) < M_EPSILON)
			{
				break;
			}
			throughput *= bsdf * abs(dot(w_i, v_z)) / pdf;
			
			ray_origin = hit_record.P + w_i * 1e-3;
			ray_direction = w_i;

			if (depth >= 2)
			{
				float q = min(max(throughput.x, max(throughput.y, throughput.z)) + 0.001f, 0.95f);
				if (rnd(seed) > q) break;
				throughput /= q;
			}
		}
	} while (--samples);

	radiance = radiance / params.sample_count;

	//temporary to reduce fireflies
	float lum = dot(radiance, make_float3(0.212671f, 0.715160f, 0.072169f));
	if (lum > 50.0f)
	{
		radiance *= 50.0f / lum;
	}

	float3 old_accum_color = params.accum_buffer[idx];
	if (params.frame_index > 0)
	{
		radiance += old_accum_color;
	}
	params.accum_buffer[idx] = radiance;
}

extern "C" 
__global__ void MISS_NAME(ms)()
{
	GetPayload<HitRecord>()->hit = false;
}

struct VertexData
{
	float3 P;
	float3 N;
	float2 uv;
};

__device__ VertexData LoadVertexData(MeshGPU const& mesh, unsigned int primitive_idx, float2 barycentrics)
{
	VertexData vertex{};
	uint3* mesh_indices = params.indices + mesh.indices_offset;

	uint3 primitive_indices = mesh_indices[primitive_idx];
	Uint32 i0 = primitive_indices.x;
	Uint32 i1 = primitive_indices.y;
	Uint32 i2 = primitive_indices.z;

	float3* mesh_vertices = params.vertices + mesh.positions_offset;
	float3 pos0 = mesh_vertices[i0];
	float3 pos1 = mesh_vertices[i1];
	float3 pos2 = mesh_vertices[i2];
	vertex.P = Interpolate(pos0, pos1, pos2, barycentrics);

	//geometric normal
	//float3 edge1 = world_v1 - world_v0;
	//float3 edge2 = world_v2 - world_v0;
	//float3 geometric_normal = normalize(cross(edge1, edge2));

	float3* mesh_normals = params.normals + mesh.normals_offset;
	float3 nor0 = mesh_normals[i0];
	float3 nor1 = mesh_normals[i1];
	float3 nor2 = mesh_normals[i2];
	vertex.N = Interpolate(nor0, nor1, nor2, barycentrics);
	
	float2* mesh_uvs = params.uvs + mesh.uvs_offset;
	float2 uv0 = mesh_uvs[i0];
	float2 uv1 = mesh_uvs[i1];
	float2 uv2 = mesh_uvs[i2];
	vertex.uv = Interpolate(uv0, uv1, uv2, barycentrics);
	vertex.uv.y = 1.0f - vertex.uv.y;
	return vertex;
}

extern "C" 
__global__ void AH_NAME(ah)()
{
	Uint32 instance_idx = optixGetInstanceId();
	Uint32 primitive_idx = optixGetPrimitiveIndex();

	MeshGPU mesh = params.meshes[instance_idx];
	VertexData vertex = LoadVertexData(mesh, optixGetPrimitiveIndex(), optixGetTriangleBarycentrics());
	MaterialGPU material = params.materials[mesh.material_idx];

	if (material.diffuse_tex_id >= 0)
	{
		float4 sampled = tex2D<float4>(params.textures[material.diffuse_tex_id], vertex.uv.x, vertex.uv.y);
		if(sampled.w < material.alpha_cutoff) optixIgnoreIntersection();
	}
}


__device__ float3 TransformVertex(float const matrix[12], float3 const& position)
{
	float3 transformed_position;
	transformed_position.x = matrix[0] * position.x + matrix[1] * position.y + matrix[2] * position.z + matrix[3];
	transformed_position.y = matrix[4] * position.x + matrix[5] * position.y + matrix[6] * position.z + matrix[7];
	transformed_position.z = matrix[8] * position.x + matrix[9] * position.y + matrix[10] * position.z + matrix[11];
	return transformed_position;
}

__device__ float3 TransformNormal(float const matrix[12], float3 const& normal)
{
	float3 transformed_normal;
	transformed_normal.x = matrix[0] * normal.x + matrix[1] * normal.y + matrix[2] * normal.z;
	transformed_normal.y = matrix[4] * normal.x + matrix[5] * normal.y + matrix[6] * normal.z;
	transformed_normal.z = matrix[8] * normal.x + matrix[9] * normal.y + matrix[10] * normal.z;
	return normalize(transformed_normal);
}

extern "C" 
__global__ void CH_NAME(ch)()
{
	MeshGPU mesh = params.meshes[optixGetInstanceId()];
	VertexData vertex = LoadVertexData(mesh, optixGetPrimitiveIndex(), optixGetTriangleBarycentrics());

	float object_to_world_transform[12];
	optixGetObjectToWorldTransformMatrix(object_to_world_transform);

	HitRecord* hit_record = GetPayload<HitRecord>();
	hit_record->hit = true;
	hit_record->P = TransformVertex(object_to_world_transform, vertex.P);
	hit_record->N = TransformNormal(object_to_world_transform, vertex.N);
	hit_record->uv = vertex.uv;
	hit_record->material_idx = mesh.material_idx;
}


extern "C"
__global__ void AH_NAME(ah_shadow)()
{
	Uint32 instance_idx = optixGetInstanceId();
	Uint32 primitive_idx = optixGetPrimitiveIndex();

	MeshGPU mesh = params.meshes[instance_idx];
	VertexData vertex = LoadVertexData(mesh, optixGetPrimitiveIndex(), optixGetTriangleBarycentrics());
	MaterialGPU material = params.materials[mesh.material_idx];

	if (material.diffuse_tex_id >= 0)
	{
		float4 sampled = tex2D<float4>(params.textures[material.diffuse_tex_id], vertex.uv.x, vertex.uv.y);
		if (sampled.w < material.alpha_cutoff) optixIgnoreIntersection();
	}
	if (material.specular_transmission > 0)
	{
		optixIgnoreIntersection();
	}
}

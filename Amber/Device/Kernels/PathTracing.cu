#include "hip/hip_runtime.h"
#pragma once
#include "DeviceCommon.cuh"
#include "Device/DeviceHostCommon.h"
#include "Disney.cuh"
#include "ONB.cuh"

using namespace amber;

extern "C" __constant__ LaunchParams params;

__device__ __forceinline__ Float3 ApplyNormalMapping(Float3 tangent_space_normal, Float3 N, Float3 T, Float3 B)
{
	return normalize(tangent_space_normal.x * T + tangent_space_normal.y * B + tangent_space_normal.z * N);
}
__device__ __forceinline__ void EvaluateMaterial(MaterialGPU const& material, EvaluatedMaterial& evaluated_material, Float2 uv)
{
	if (material.diffuse_tex_id >= 0)
	{
		Float4 sampled = tex2D<Float4>(params.textures[material.diffuse_tex_id], uv.x, uv.y);
		evaluated_material.base_color = material.base_color * MakeFloat3(sampled.x, sampled.y, sampled.z);
	}
	else
	{
		evaluated_material.base_color = material.base_color;
	}

	if (material.emissive_tex_id >= 0)
	{
		Float4 sampled = tex2D<Float4>(params.textures[material.emissive_tex_id], uv.x, uv.y);
		evaluated_material.emissive = material.emissive_color * MakeFloat3(sampled.x, sampled.y, sampled.z);
	}
	else
	{
		evaluated_material.emissive = material.emissive_color;
	}

	if (material.metallic_roughness_tex_id >= 0)
	{
		Float4 sampled = tex2D<Float4>(params.textures[material.metallic_roughness_tex_id], uv.x, uv.y);
		evaluated_material.ao = sampled.x;
		evaluated_material.roughness = sampled.y * material.roughness;
		evaluated_material.metallic = sampled.z * material.metallic;
	}
	else
	{
		evaluated_material.ao = 1.0f;
		evaluated_material.roughness = material.roughness;
		evaluated_material.metallic = material.metallic;
	}

	if (material.normal_tex_id >= 0)
	{
		Float4 sampled = tex2D<Float4>(params.textures[material.normal_tex_id], uv.x, uv.y);
		evaluated_material.tangent_space_normal = MakeFloat3(sampled.x, sampled.y, sampled.z);
		evaluated_material.tangent_space_normal = 2.0f * evaluated_material.tangent_space_normal - 1.0f;
	}
	else
	{
		evaluated_material.tangent_space_normal = MakeFloat3(0.0f, 0.0f, 1.0f);
	}

	evaluated_material.specular_tint = material.specular_tint;
	evaluated_material.anisotropy = material.anisotropy;
	evaluated_material.sheen = material.sheen;
	evaluated_material.sheen_tint = material.sheen_tint;
	evaluated_material.clearcoat = material.clearcoat;
	evaluated_material.clearcoat_gloss = material.clearcoat_gloss;
	evaluated_material.ior = material.ior;
	evaluated_material.specular_transmission = material.specular_transmission;
}
__device__ __forceinline__ Float3 GetRayDirection(Uint2 pixel, Uint2 screen, PRNG& prng)
{
	Float3 const U = params.cam_u;
	Float3 const V = params.cam_v;
	Float3 const W = params.cam_w;

	Float2 subpixelJitter = prng.RandomFloat2();
	Float2 d = (MakeFloat2(pixel) + subpixelJitter) / MakeFloat2(screen);
	d.y = 1.0f - d.y;
	d = 2.0f * d - 1.0f;
	Float tanHalfFovy = tan(params.cam_fovy * 0.5f);
	Float aspectRatio = params.cam_aspect_ratio;
	Float3 ray_direction = normalize(d.x * aspectRatio * tanHalfFovy * U + d.y * tanHalfFovy * V + W);
	return ray_direction;
}

__device__ __forceinline__ ColorRGB32F SampleDirectLight(EvaluatedMaterial const& evaluated_material, Float3 const& hit_point, Float3 const& wo, 
	Float3 const& T, Float3 const& B, Float3 const& N, PRNG& prng)
{
	Uint32 light_index = prng.RandomFloat() * params.light_count;
	LightGPU light = params.lights[light_index];

	ColorRGB32F radiance(0.0f);
	if (light.type == LightGPUType_Directional)
	{
		Float3 light_direction = normalize(light.direction);
		if (!TraceOcclusion(params.traversable, hit_point + M_EPSILON * N, -light_direction, M_EPSILON, M_INF))
		{
			ColorRGB32F bsdf = DisneyBrdf(evaluated_material, N, wo, -light_direction, T, B);
			radiance = bsdf * light.color * abs(dot(-light_direction, N));
		}

		Float3 wi;
		Float bsdf_pdf;
		ColorRGB32F bsdf = SampleDisneyBrdf(evaluated_material, N, wo, T, B, prng, wi, bsdf_pdf);
		if (bsdf.Length() > M_EPSILON && bsdf_pdf >= M_EPSILON)
		{
			Float light_pdf = 1.0f; 
			Float w = PowerHeuristic(1.f, bsdf_pdf, 1.f, light_pdf);

			if (!TraceOcclusion(params.traversable, hit_point + M_EPSILON * N, wi, M_EPSILON, M_INF))
			{
				ColorRGB32F bsdf = DisneyBrdf(evaluated_material, N, wo, -light_direction, T, B);
				radiance += bsdf * light.color * abs(dot(wi, N)) * w / bsdf_pdf;
			}
		}
	}
	else if (light.type == LightGPUType_Point)
	{
		Float3 light_pos = light.position;
		Float3 light_dir = light_pos - hit_point; 
		Float dist = length(light_dir);
		light_dir = light_dir / dist;

		if (!TraceOcclusion(params.traversable, hit_point + M_EPSILON * N, light_dir, M_EPSILON, dist - M_EPSILON))
		{
			Float attenuation = 1.0f / (dist * dist);
			ColorRGB32F bsdf = DisneyBrdf(evaluated_material, N, wo, light_dir, T, B);
			radiance = bsdf * light.color * abs(dot(light_dir, N)) * attenuation;
		}

		Float3 w_i;
		Float bsdf_pdf;
		ColorRGB32F bsdf = SampleDisneyBrdf(evaluated_material, N, wo, T, B, prng, w_i, bsdf_pdf);
		if (bsdf.Length() > M_EPSILON && bsdf_pdf >= M_EPSILON)
		{
			Float light_pdf = (dist * dist) / (abs(dot(light_dir, N)) * 1.0f); // light.radius);
			Float w = PowerHeuristic(1.f, bsdf_pdf, 1.f, light_pdf);

			if (!TraceOcclusion(params.traversable, hit_point + M_EPSILON * N, w_i, M_EPSILON, dist - M_EPSILON))
			{
				Float attenuation = 1.0f / (dist * dist); 
				radiance += bsdf * light.color * abs(dot(w_i, N)) * attenuation * w / bsdf_pdf;
			}
		}
	}
	return radiance;
}

__device__ __forceinline__ void WriteToDenoiserBuffers(Uint32 idx, Float3 const& albedo, Float3 const& normal)
{
	if (params.denoiser_albedo != NULL)
	{
		params.denoiser_albedo[idx] = albedo;
	}

	if (params.denoiser_normals != NULL)
	{
		Float3 view_normal;
		view_normal.x = dot(normal,  params.cam_u);
		view_normal.y = dot(normal,  params.cam_v);
		view_normal.z = dot(normal, -params.cam_w);
		params.denoiser_normals[idx] = view_normal;
	}
}
__device__ __forceinline__ void WriteToDebugBuffer(Uint32 idx, Float3 const& albedo, Float3 const& normal, Float2 const& uv, Uint32 material_id)
{
	if (params.output_type == PathTracerOutputGPU_Albedo)
	{
		params.debug_buffer[idx] = albedo;
		return;
	}
	if (params.output_type == PathTracerOutputGPU_Normal)
	{
		params.debug_buffer[idx] = normal;
		return;
	}
	if (params.output_type == PathTracerOutputGPU_UV)
	{
		params.debug_buffer[idx] = MakeFloat3(uv, 0.0f);
		return;
	}
	if(params.output_type == PathTracerOutputGPU_MaterialID)
	{
		Float3 material_id_color = MakeFloat3(
			(material_id * 37) % 255 / 255.0, 
			(material_id * 59) % 255 / 255.0,
			(material_id * 97) % 255 / 255.0);
		params.debug_buffer[idx] = material_id_color;
		return;
	}
}


extern "C" __global__ void RG_NAME(rg)()
{
	OptixTraversableHandle scene = params.traversable;
	Float3 const  eye = params.cam_eye;
	Uint2  const  pixel  = MakeUint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);
	Uint2  const  screen = MakeUint2(optixGetLaunchDimensions().x, optixGetLaunchDimensions().y);
	Uint32 samples = params.sample_count;
	Uint32 idx = pixel.x + pixel.y * screen.x;

	ColorRGB32F radiance(0.0f);
	ColorRGB32F throughput(1.0f);
	do
	{
		PRNG prng = PRNG::Create(idx, samples + params.frame_index);
		Float3 ray_origin = eye;
		Float3 ray_direction = GetRayDirection(pixel, screen, prng);

		HitRecord hit_record{};
		Uint32 p0 = PackPointer0(&hit_record), p1 = PackPointer1(&hit_record);

		for (Uint32 depth = 0; depth < params.max_depth; ++depth)
		{
			Trace(scene, ray_origin, ray_direction, M_EPSILON, M_INF, p0, p1);
			if (!hit_record.hit)
			{
				Float3 const& dir = ray_direction;
				Float u = (1.f + atan2(dir.x, -dir.z) * M_INV_PI) * 0.5f;
				Float v = 1.0f - acos(dir.y) * M_INV_PI;
				Float3 env_map_color = MakeFloat3(0.0f);
				if (params.sky)
				{
					Float4 sampled = tex2D<Float4>(params.sky, u, v);
					env_map_color = MakeFloat3(sampled.x, sampled.y, sampled.z);
				}
				radiance += env_map_color * throughput;

				if (depth == 0)
				{
					WriteToDenoiserBuffers(idx, MakeFloat3(0.0f, 0.0f, 0.0f), MakeFloat3(0.0f, 0.0f, 0.0f));
					WriteToDebugBuffer(idx, MakeFloat3(0.0f, 0.0f, 0.0f), MakeFloat3(0.0f, 0.0f, 0.0f), MakeFloat2(0.0f, 0.0f), 0);
				}
				break;
			}

			MeshGPU mesh = params.meshes[hit_record.instance_idx];
			MaterialGPU material_gpu = params.materials[mesh.material_idx];

			EvaluatedMaterial material{};
			EvaluateMaterial(material_gpu, material, hit_record.uv);

			ColorRGB32F emissive = material.emissive;
			radiance += emissive * throughput;

			Float3 w_o = -ray_direction;
			Float3 T, B;
			Float3 Ns = hit_record.Ns;
			Float3 Ng = hit_record.Ng;
			if (material.specular_transmission == 0.0f && dot(w_o, Ns) < 0.0f)
			{
				Ns = -Ns;
				Ng = -Ng;
			}
			BuildONB(Ns, T, B);
			Ns = ApplyNormalMapping(material.tangent_space_normal, Ns, T, B);

			if (depth == 0)
			{
				WriteToDenoiserBuffers(idx, (Float3)material.base_color, Ns);
				WriteToDebugBuffer(idx, (Float3)material.base_color, Ng, hit_record.uv, mesh.material_idx);
			}

			radiance += SampleDirectLight(material, hit_record.P, w_o, T, B, Ns, prng) * throughput;

			Float3 w_i;
			Float pdf;
			ColorRGB32F bsdf = SampleDisneyBrdf(material, Ns, w_o, T, B, prng, w_i, pdf);
			if (params.output_type == PathTracerOutputGPU_Custom && depth == 0)
			{
				Bool entering = dot(w_o, Ns) > 0.0f;
				Float dot_wi_n = dot(w_i, Ns); 
				Bool is_reflected = SameHemisphere(w_o, w_i, Ns); 
				params.debug_buffer[idx] = MakeFloat3(is_reflected, is_reflected, is_reflected);
				return;
			}

			if (pdf == 0.0f || bsdf.Length() < M_EPSILON)
			{
				break;
			}
			throughput *= bsdf * abs(dot(w_i, Ns)) / pdf;
			
			ray_origin = hit_record.P + w_i * 1e-3;
			ray_direction = w_i;

			if (depth >= 2)
			{
				Float q = min(max(throughput.r, max(throughput.g, throughput.b)) + 0.001f, 0.95f);
				if (prng.RandomFloat() > q) break;
				throughput /= q;
			}
		}
	} while (--samples);

	radiance = radiance / params.sample_count;

	Float luminance = radiance.Luminance(); 
	if (luminance > 50.0f)
	{
		radiance *= 50.0f / luminance;
	}
	Float3 old_accum_color = params.accum_buffer[idx];
	if (params.frame_index > 0)
	{
		radiance += old_accum_color;
	}
	params.accum_buffer[idx] = static_cast<Float3>(radiance);
}

extern "C" __global__ void MISS_NAME(ms)()
{
	GetPayload<HitRecord>()->hit = false;
}

struct VertexData
{
	Float3 P;
	Float3 Ng;
	Float3 Ns;
	Float2 uv;
};
__device__ VertexData LoadVertexData(MeshGPU const& mesh, Uint32 primitive_idx, Float2 barycentrics)
{
	VertexData vertex{};
	Uint3* mesh_indices = params.indices + mesh.indices_offset;

	Uint3 primitive_indices = mesh_indices[primitive_idx];
	Uint32 i0 = primitive_indices.x;
	Uint32 i1 = primitive_indices.y;
	Uint32 i2 = primitive_indices.z;

	Float3* mesh_vertices = params.vertices + mesh.positions_offset;
	Float3 pos0 = mesh_vertices[i0];
	Float3 pos1 = mesh_vertices[i1];
	Float3 pos2 = mesh_vertices[i2];
	vertex.P = Interpolate(pos0, pos1, pos2, barycentrics);

	Float3 edge1 = pos1 - pos0;
	Float3 edge2 = pos2 - pos0;
	vertex.Ng = normalize(cross(edge1, edge2));

	Float3* mesh_normals = params.normals + mesh.normals_offset;
	Float3 nor0 = mesh_normals[i0];
	Float3 nor1 = mesh_normals[i1];
	Float3 nor2 = mesh_normals[i2];
	vertex.Ns = Interpolate(nor0, nor1, nor2, barycentrics);
	
	Float2* mesh_uvs = params.uvs + mesh.uvs_offset;
	Float2 uv0 = mesh_uvs[i0];
	Float2 uv1 = mesh_uvs[i1];
	Float2 uv2 = mesh_uvs[i2];
	vertex.uv = Interpolate(uv0, uv1, uv2, barycentrics);
	vertex.uv.y = 1.0f - vertex.uv.y;
	return vertex;
}

extern "C"  __global__ void AH_NAME(ah)()
{
	Uint32 instance_idx = optixGetInstanceId();
	Uint32 primitive_idx = optixGetPrimitiveIndex();

	MeshGPU mesh = params.meshes[instance_idx];
	MaterialGPU material = params.materials[mesh.material_idx];
	if (material.diffuse_tex_id >= 0)
	{
		VertexData vertex = LoadVertexData(mesh, primitive_idx, optixGetTriangleBarycentrics());
		Float4 sampled = tex2D<Float4>(params.textures[material.diffuse_tex_id], vertex.uv.x, vertex.uv.y);
		if (sampled.w < material.alpha_cutoff)
		{
			optixIgnoreIntersection();
		}
	}
}

__device__ Float3 TransformVertex(Float const matrix[12], Float3 const& position)
{
	Float3 transformed_position;
	transformed_position.x = matrix[0] * position.x + matrix[1] * position.y + matrix[2] * position.z + matrix[3];
	transformed_position.y = matrix[4] * position.x + matrix[5] * position.y + matrix[6] * position.z + matrix[7];
	transformed_position.z = matrix[8] * position.x + matrix[9] * position.y + matrix[10] * position.z + matrix[11];
	return transformed_position;
}
__device__ Float3 TransformNormal(Float const matrix[12], Float3 const& normal)
{
	Float3 transformed_normal;
	transformed_normal.x = matrix[0] * normal.x + matrix[1] * normal.y + matrix[2] * normal.z;
	transformed_normal.y = matrix[4] * normal.x + matrix[5] * normal.y + matrix[6] * normal.z;
	transformed_normal.z = matrix[8] * normal.x + matrix[9] * normal.y + matrix[10] * normal.z;
	return normalize(transformed_normal);
}

extern "C" __global__ void CH_NAME(ch)()
{
	Uint32 instance_idx = optixGetInstanceId();
	Uint32 primitive_idx = optixGetPrimitiveIndex();
	Float2 barycentrics = optixGetTriangleBarycentrics();

	MeshGPU mesh = params.meshes[instance_idx];
	MaterialGPU material = params.materials[mesh.material_idx];
	VertexData vertex = LoadVertexData(mesh, primitive_idx, barycentrics);

	Float object_to_world_transform[12];
	optixGetObjectToWorldTransformMatrix(object_to_world_transform);

	HitRecord* hit_record = GetPayload<HitRecord>();
	hit_record->P = TransformVertex(object_to_world_transform, vertex.P);
	hit_record->Ng = TransformNormal(object_to_world_transform, vertex.Ng);
	hit_record->uv = vertex.uv;
	hit_record->barycentrics = barycentrics;
	hit_record->primitive_idx = primitive_idx;
	hit_record->instance_idx = instance_idx;
	hit_record->hit = true;
	hit_record->t = optixGetRayTmax();
	hit_record->Ns = TransformNormal(object_to_world_transform, vertex.Ns);
}

extern "C" __global__ void AH_NAME(ah_shadow)()
{
	Uint32 instance_idx = optixGetInstanceId();
	Uint32 primitive_idx = optixGetPrimitiveIndex();
	Float2 barycentrics = optixGetTriangleBarycentrics();

	MeshGPU mesh = params.meshes[instance_idx];
	MaterialGPU material = params.materials[mesh.material_idx];
	if (material.diffuse_tex_id >= 0)
	{
		VertexData vertex = LoadVertexData(mesh, primitive_idx, barycentrics);
		Float4 sampled = tex2D<Float4>(params.textures[material.diffuse_tex_id], vertex.uv.x, vertex.uv.y);
		if (sampled.w < material.alpha_cutoff)
		{
			optixIgnoreIntersection();
		}
	}
	if (material.specular_transmission > 0)
	{
		optixIgnoreIntersection();
	}
}


/*
__device__ __forceinline__ Float3 ApplyNormalMapping(HitRecord const& hit_record)
{
	MeshGPU const& mesh = params.meshes[hit_record.instance_idx];
	Uint32 primitive_idx = hit_record.primitive_idx;
	Float2 barycentrics = hit_record.barycentrics;
	Float2 uv = hit_record.uv;
	Float3 N = hit_record.Ns;
	MaterialGPU material_gpu = params.materials[mesh.material_idx];
	if (material_gpu.normal_tex_id < 0)
	{
		return N;
	}
	Float4 sampled = tex2D<Float4>(params.textures[material_gpu.normal_tex_id], uv.x, uv.y);
	Float3 normal_tangent_space = MakeFloat3(sampled.x, sampled.y, sampled.z);
	normal_tangent_space = 2.0f * normal_tangent_space - 1.0f;

	MaterialGPU material = params.materials[mesh.material_idx];
	Uint3* mesh_indices = params.indices + mesh.indices_offset;

	Uint3 primitive_indices = mesh_indices[primitive_idx];
	Uint32 i0 = primitive_indices.x;
	Uint32 i1 = primitive_indices.y;
	Uint32 i2 = primitive_indices.z;

	Float2* mesh_uvs = params.uvs + mesh.uvs_offset;
	Float2 uv0 = mesh_uvs[i0];
	Float2 uv1 = mesh_uvs[i1];
	Float2 uv2 = mesh_uvs[i2];

	Float2 deltaUV_10 = uv1 - uv0;
	Float2 deltaUV_20 = uv2 - uv0;

	Float3* mesh_vertices = params.vertices + mesh.positions_offset;
	Float3 P0 = mesh_vertices[i0];
	Float3 P1 = mesh_vertices[i1];
	Float3 P2 = mesh_vertices[i2];

	Float3 edge_P0P1 = P1 - P0;
	Float3 edge_P0P2 = P2 - P0;

	float det_inverse = 1.0f / (deltaUV_10.x * deltaUV_20.y - deltaUV_10.y * deltaUV_20.x);
	float3 T = (edge_P0P1 * deltaUV_20.y - edge_P0P2 * deltaUV_10.y) * det_inverse;
	float3 B = (edge_P0P2 * deltaUV_10.x - edge_P0P1 * deltaUV_20.x) * det_inverse;
	return LocalToWorldFrame(T, B, N, normal_tangent_space);
}*/

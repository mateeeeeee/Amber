#include "hip/hip_runtime.h"
#pragma once
#include <optix.h>
#include "OptixShared.h"
#include "CudaMath.h"

using namespace lavender;

extern "C" 
{
	__constant__ Params params;
}

__forceinline__ __device__ float3 ToSRGB(float3 const& color)
{
	static constexpr float INV_GAMMA = 1.0f / 2.2f;
	float3 gamma_corrected_color = make_float3(powf(color.x, INV_GAMMA), powf(color.y, INV_GAMMA), powf(color.z, INV_GAMMA));
	return make_float3(
		color.x < 0.0031308f ? 12.92f * color.x : 1.055f * gamma_corrected_color.x - 0.055f,
		color.y < 0.0031308f ? 12.92f * color.y : 1.055f * gamma_corrected_color.y - 0.055f,
		color.z < 0.0031308f ? 12.92f * color.z : 1.055f * gamma_corrected_color.z - 0.055f);
}
__forceinline__ __device__ unsigned char QuantizeUnsigned8Bits(float x)
{
	x = clamp(x, 0.0f, 1.0f);
	enum { N = (1 << 8) - 1, Np1 = (1 << 8) };
	return (unsigned char)min((unsigned int)(x * (float)Np1), (unsigned int)N);
}
__forceinline__ __device__ uchar4 MakeColor(const float3& c)
{
	float3 srgb = ToSRGB(c);
	return make_uchar4(QuantizeUnsigned8Bits(srgb.x), QuantizeUnsigned8Bits(srgb.y), QuantizeUnsigned8Bits(srgb.z), 255u);
}

__forceinline__ __device__ void SetPayload(float3 p)
{
	optixSetPayload_0(__float_as_uint(p.x));
	optixSetPayload_1(__float_as_uint(p.y));
	optixSetPayload_2(__float_as_uint(p.z));
}
__forceinline__ __device__ float3 GetPayload(unsigned int p0, unsigned int p1, unsigned int p2)
{
	float3 p;
	p.x = __uint_as_float(p0);
	p.y = __uint_as_float(p1);
	p.z = __uint_as_float(p2);
	return p;
}

static __forceinline__ __device__ void computeRay(uint3 idx, uint3 dim, float3& origin, float3& direction)
{
	float3 U = params.cam_u;
	float3 V = params.cam_v;
	float3 W = params.cam_w;
	float2 d = 2.0f * make_float2((float)idx.x / dim.x, (float)idx.y / dim.y) - 1.0f;

	const float tanFovyHalf = tan(params.cam_fovy * 0.5f);
	const float aspectRatio = params.cam_aspect_ratio;

	origin = params.cam_eye;
	direction = normalize((d.x * aspectRatio * tanFovyHalf) * U + (d.y * tanFovyHalf) * V + W);
}



extern "C" __global__ void RG_NAME(rg)()
{
	const uint3 idx = optixGetLaunchIndex();
	const uint3 dim = optixGetLaunchDimensions();

	float3 rayOrigin, rayDirection;
	computeRay(idx, dim, rayOrigin, rayDirection);

	unsigned int p0, p1, p2;
	optixTrace(
		params.handle,
		rayOrigin,
		rayDirection,
		0.0f,						
		1e16f,						
		0.0f,						
		OptixVisibilityMask(255),	
		OPTIX_RAY_FLAG_NONE,
		0,                   
		1,                   
		0,                   
		p0, p1, p2);
	float3 result = GetPayload(p0, p1, p2);
	params.image[idx.y * params.image_width + idx.x] = MakeColor(result);
}


extern "C" __global__ void __miss__ms()
{
	MissData* miss_data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
	SetPayload(make_float3(0.0f, 0.0f, 1.0f));
}


extern "C" __global__ void __closesthit__ch()
{
	//const float2 barycentrics = optixGetTriangleBarycentrics();
	SetPayload(make_float3(0.0f, 1.0f, 0.0f));
}

